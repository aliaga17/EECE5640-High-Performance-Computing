
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define n 32

__global__ void stencil(float *a, float *b, long N){
    //Global thread identifier
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    //Weird indexing to get the correct mapping between thread id and i, j, k
    int element_idx = (thread_id / 6);
    int i = element_idx % (n-2) + 1;
    int j = (element_idx - i + 1) / (n-2) % (n-2) + 1;
    int k = ((element_idx - i + 1) / (n-2) - j + 1) / (n-2) % (n-2) + 1;
    float tmp;

    //Using as many threads as operations, not more
    if(thread_id < N){
        switch(thread_id % 6){
            case 0:
                tmp = 0.8 * b[k * n * n + j * n + (i - 1)];
                atomicAdd(&a[k * n * n + j * n + i], tmp);
                break;
            case 1:
                tmp = 0.8 * b[k * n * n + j * n + (i + 1)];
                atomicAdd(&a[k * n * n + j * n + i], tmp);
                break;
            case 2:
                tmp = 0.8 * b[k * n * n + (j - 1) * n + i];
                atomicAdd(&a[k * n * n + j * n + i], tmp);
                break;
            case 3:
                tmp = 0.8 * b[k * n * n + (j + 1) * n + i];
                atomicAdd(&a[k * n * n + j * n + i], tmp);
                break;
            case 4:
                tmp = 0.8 * b[(k - 1) * n * n + j * n + i];
                atomicAdd(&a[k * n * n + j * n + i], tmp);
                break;
            case 5:
                tmp = 0.8 * b[(k + 1) * n * n + j * n + i];
                atomicAdd(&a[k * n * n + j * n + i], tmp);
                break;
        }
    }

}

int main(int argc, char *argv[])
{
    //timing variables
    struct timeval  start, end;
    double comp_time;

    //Start timer
    gettimeofday(&start, NULL);

    //Memory allocation
    float* a;
    float* b;
    size_t bytes = (n * n * n) * sizeof(float);
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);

    //Initialize b
    for (int i=0; i<(n * n * n); i++) b[i] = 1;

    //Set dimensinos of blocks and grid
    long N = (n-2) * (n-2) * (n-2) * 6; //Total number of operations for the 6-point 3d stencil
    int THREADS = 512;
    int BLOCKS = (N + THREADS - 1) / THREADS;

    stencil<<<BLOCKS, THREADS>>>(a, b, N);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    hipDeviceSynchronize();

    //Stop timer
    gettimeofday(&end, NULL);
    comp_time = ((double) ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec))); 
    printf("Total computing time for n=%d: %f\n",n, comp_time);
    printf("Total number of threads = %d\n", N);

    //Testing for small n
    /*printf("RESULTS a:\n");
    for (int k=0; k<n; k++){
        for (int j=0; j<n; j++){
            for (int i=0; i<n; i++) {
                printf("%.1f  ", a[k * n * n + j * n + i]);
            }
            printf("\n");
        }
        printf("\n");
    }*/

    hipFree(a);
    hipFree(b);

    return 0;
}
