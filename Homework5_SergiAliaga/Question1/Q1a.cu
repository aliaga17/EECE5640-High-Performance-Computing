//Program to compute an histogram using cuda kernels

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define N (1 << 10) 
#define UPPER_LIM 10000000
#define LOWER_LIM  1
#define NBINS 100

/* generate random numbers within the specified limit */
int generate_random_number(unsigned int lower_limit, unsigned int upper_limit) {
    return lower_limit + (upper_limit - lower_limit) * ((double)rand() / RAND_MAX);
}

__global__ void histogram(int *data, int *bins, int n_data, int n_bins, int DIV, int *results){
    //Global thread identifier
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    //Using as many threads as data input elements, not more
    if(thread_id < n_data){
        int bin = (data[thread_id] - LOWER_LIM)/ DIV;
        atomicAdd(&bins[bin], 1);
        if(results[bin] == 0) results[bin] = data[thread_id];
    }
}

int main(int argc, char *argv[]){

    //timing variables
    struct timeval  start, end;
    double comp_time;

    //Start timer
    gettimeofday(&start, NULL);

    //Data 
    size_t bytes_data =N * sizeof(int);
    //Bins
    size_t bytes_bins = NBINS * sizeof(int);
    
    //Memory allocation
    int *data;
    int *bins;
    //Auxiliar data structure to print one element of each class
    int *results;
    hipMallocManaged(&data, bytes_data);
    hipMallocManaged(&bins, bytes_bins);
    hipMallocManaged(&results, bytes_bins);
 
    //Initialize data with random numbers
    for (int i = 0; i < N; i ++) {
        data[i] = generate_random_number(LOWER_LIM, UPPER_LIM);
    }
    //Set divisor for finding the corresponding bin for an input, 
    // accounting for bins of irregular size in case UPPER_LIM - LOWER_LIM is not 
    // divisible by NBINS
    int DIV = (UPPER_LIM - LOWER_LIM + NBINS - 1) / NBINS;

    //Initialize bins and result
    for(int i = 0; i<NBINS; i++){
        bins[i] = 0;
        results[i] = 0;
    }

    //Set dimensions of blocks and grid
    int THREADS = 512;
    int BLOCKS = (N + THREADS - 1) / THREADS;

    //printf("threads:%d\n", THREADS);
    //printf("Blocks:%d\n", BLOCKS);

    histogram<<<BLOCKS, THREADS>>>(data, bins, N, NBINS, DIV, results);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
        
    hipDeviceSynchronize();

    //Stop timer
    gettimeofday(&end, NULL);
    comp_time = ((double) ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec))); 
    printf("Total computing time: %f\n", comp_time);

    printf("Elements of each class of the %d bins:\n", NBINS);
    int tmp = 0;
    for(int i = 0; i<NBINS; i++){
        tmp+=bins[i];
        printf("%d, ", results[i]);
    }
    printf("\nTotal number of elements counted: %d\n", tmp);

    hipFree(data);
    hipFree(bins);
    hipFree(results);

    return 0;
}
