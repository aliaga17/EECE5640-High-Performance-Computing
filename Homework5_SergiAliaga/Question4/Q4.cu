#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

// Approximation of the number PI through the Leibniz's series
// Language: C

//atomicAdd version for double precision
#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

__global__ void computePi(double *pi, double n){
    //Global thread identifier
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    float i = 2 * thread_id + 1;
    int s = (1-2*(thread_id%2));
    double tmp;

    //Using as many threads as operations, not more
    if(i <= (2 * n)){
        tmp = s * (4 / i);
        //atomicAdd(pi, tmp);
    }

}

int main(int argc, char *argv[])
{
    //timing variables
    struct timeval  start, end;
    double comp_time;

    //Start timer
    gettimeofday(&start, NULL);

    double n;// Number of iterations and control variable

    printf("GPU implementation of the approximation of the number PI through the Leibniz's series\n");
    printf("\nEnter the number of iterations: ");    
    scanf("%lf",&n);
    printf("\nPlease wait. Running...\n");    

    double* pi;
    hipMallocManaged(&pi, sizeof(double));
    *pi = 0.0;

    //Set dimensinos of blocks and grid
    int THREADS = 512;
    int BLOCKS = (n + THREADS - 1) / THREADS;

    computePi<<<BLOCKS, THREADS>>>(pi, n);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    hipDeviceSynchronize();

    //Stop timer
    gettimeofday(&end, NULL);
    comp_time = ((double) ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec))); 
    printf("\nAproximated value of PI = %1.16lf\n", *pi);
    printf("DOUBLE precision total computing time for n=%.0lf: %f\n", n, comp_time);

    hipFree(pi);

    return 0;
}