
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

// Approximation of the number PI through the Leibniz's series
// Language: C

__global__ void computePi(float *pi, double n){
    //Global thread identifier
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    float i = 2 * thread_id + 1;
    int s = (1-2*(thread_id%2));
    float tmp;

    //Using as many threads as operations, not more
    if(i <= (2 * n)){
        tmp = s * (4 / i);
        atomicAdd(pi, tmp);
    }

}

int main(int argc, char *argv[])
{
    //timing variables
    struct timeval  start, end;
    double comp_time;

    //Start timer
    gettimeofday(&start, NULL);

    double n;// Number of iterations and control variable

    printf("GPU implementation of the approximation of the number PI through the Leibniz's series\n");
    printf("\nEnter the number of iterations: ");    
    scanf("%lf",&n);
    printf("\nPlease wait. Running...\n");    

    float* pi;
    hipMallocManaged(&pi, sizeof(float));
    *pi = 0.0;

    //Set dimensinos of blocks and grid
    int THREADS = 512;
    int BLOCKS = (n + THREADS - 1) / THREADS;

    computePi<<<BLOCKS, THREADS>>>(pi, n);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    hipDeviceSynchronize();

    //Stop timer
    gettimeofday(&end, NULL);
    comp_time = ((double) ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec))); 
    printf("\nAproximated value of PI = %1.16lf\n", *pi);
    printf("SINGLE precision total computing time for n=%.0lf: %f\n", n, comp_time);

    hipFree(pi);

    return 0;
}